// #include "grid.cuh"
#include "/home/a2rlab4/GRiDBenchmarks/GRiD/grid.cuh"
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>


int iiwa(){
    grid::gridData<float> *hd_data = grid::init_gridData<float,1>();
    grid::robotModel<float> *d_robotModel = grid::init_robotModel<float>();;
    const int num_timesteps = 1;
    float gravity = static_cast<float>(9.81);
    dim3 dimms(grid::SUGGESTED_THREADS,1,1);
    hipStream_t *streams = grid::init_grid<float>();
    hd_data->h_q_qd_u[0] = 1.24;
    hd_data->h_q_qd_u[1] = 0.13;
    hd_data->h_q_qd_u[2] = -0.17;
    hd_data->h_q_qd_u[3] = 1.33;
    hd_data->h_q_qd_u[4] = 0.22;
    hd_data->h_q_qd_u[5] = -0.56;
    hd_data->h_q_qd_u[6] = 0.99;

    hd_data->h_q_qd_u[7] = 0;
    hd_data->h_q_qd_u[8] = 0;
    hd_data->h_q_qd_u[9] = 0;
    hd_data->h_q_qd_u[10] = 0;
    hd_data->h_q_qd_u[11] = 0;
    hd_data->h_q_qd_u[12] = 0;
    hd_data->h_q_qd_u[13] = 0;

    hd_data->h_q_qd_u[14] = 0;
    hd_data->h_q_qd_u[15] = 0;
    hd_data->h_q_qd_u[16] = 0;
    hd_data->h_q_qd_u[17] = 0;
    hd_data->h_q_qd_u[18] = 0;
    hd_data->h_q_qd_u[19] = 0;
    hd_data->h_q_qd_u[20] = 0;

    gpuErrchk(hipMemcpy(hd_data->d_q_qd_u,hd_data->h_q_qd_u,3*grid::NUM_JOINTS*sizeof(float),hipMemcpyHostToDevice));
    gpuErrchk(hipDeviceSynchronize());

    printf("q,qd,u\n");
    printMat<float,1,grid::NUM_JOINTS>(hd_data->h_q_qd_u,1);
    printMat<float,1,grid::NUM_JOINTS>(&hd_data->h_q_qd_u[grid::NUM_JOINTS],1);
    printMat<float,1,grid::NUM_JOINTS>(&hd_data->h_q_qd_u[2*grid::NUM_JOINTS],1);

    printf("aba\n");
    grid::aba<float>(hd_data, d_robotModel, gravity, 1, dim3(1,1,1), dimms, streams);
    printMat<float,1,grid::NUM_JOINTS>(hd_data->h_qdd,1);
    return 0;
}

int hyq(){
    grid::gridData<float> *hd_data = grid::init_gridData<float,1>();
    grid::robotModel<float> *d_robotModel = grid::init_robotModel<float>();;
    const int num_timesteps = 1;
    float gravity = static_cast<float>(9.81);
    dim3 dimms(grid::SUGGESTED_THREADS,1,1);
    hipStream_t *streams = grid::init_grid<float>();
    hd_data->h_q_qd_u[0] = 0.8;
    hd_data->h_q_qd_u[1] = 0.3;
    hd_data->h_q_qd_u[2] = 1;
    hd_data->h_q_qd_u[3] = 0.2;
    hd_data->h_q_qd_u[4] = 0.7;
    hd_data->h_q_qd_u[5] = 0.6;
    hd_data->h_q_qd_u[6] = 0.4;
    hd_data->h_q_qd_u[7] = 0.5;
    hd_data->h_q_qd_u[8] = 0.3;
    hd_data->h_q_qd_u[9] = 0.8;
    hd_data->h_q_qd_u[10] = 0.1;
    hd_data->h_q_qd_u[11] = 0.2;

    hd_data->h_q_qd_u[12] = 0.1;
    hd_data->h_q_qd_u[13] = 0.1;
    hd_data->h_q_qd_u[14] = 0.1;
    hd_data->h_q_qd_u[15] = 0.1;
    hd_data->h_q_qd_u[16] = 0.1;
    hd_data->h_q_qd_u[17] = 0.1;
    hd_data->h_q_qd_u[18] = 0.1;
    hd_data->h_q_qd_u[19] = 0.1;
    hd_data->h_q_qd_u[20] = 0.1;
    hd_data->h_q_qd_u[21] = 0.1;
    hd_data->h_q_qd_u[22] = 0.1;
    
    hd_data->h_q_qd_u[23] = 0.1;
    hd_data->h_q_qd_u[24] = 0.1;
    hd_data->h_q_qd_u[25] = 0.1;
    hd_data->h_q_qd_u[26] = 0.1;
    hd_data->h_q_qd_u[27] = 0.1;
    hd_data->h_q_qd_u[28] = 0.1;
    hd_data->h_q_qd_u[29] = 0.1;
    hd_data->h_q_qd_u[30] = 0.1;
    hd_data->h_q_qd_u[31] = 0.1;
    hd_data->h_q_qd_u[32] = 0.1;


    gpuErrchk(hipMemcpy(hd_data->d_q_qd_u,hd_data->h_q_qd_u,3*grid::NUM_JOINTS*sizeof(float),hipMemcpyHostToDevice));
    gpuErrchk(hipDeviceSynchronize());

    printf("q,qd,u\n");
    printMat<float,1,grid::NUM_JOINTS>(hd_data->h_q_qd_u,1);
    printMat<float,1,grid::NUM_JOINTS>(&hd_data->h_q_qd_u[grid::NUM_JOINTS],1);
    printMat<float,1,grid::NUM_JOINTS>(&hd_data->h_q_qd_u[2*grid::NUM_JOINTS],1);

    printf("aba\n");
    grid::aba<float>(hd_data, d_robotModel, gravity, 1, dim3(1,1,1), dimms, streams);
    printMat<float,1,grid::NUM_JOINTS>(hd_data->h_qdd,1);
    
    return 0;
}

int hyq_missing_limb(){
    grid::gridData<float> *hd_data = grid::init_gridData<float,1>();
    grid::robotModel<float> *d_robotModel = grid::init_robotModel<float>();;
    const int num_timesteps = 1;
    float gravity = static_cast<float>(9.81);
    dim3 dimms(grid::SUGGESTED_THREADS,1,1);
    hipStream_t *streams = grid::init_grid<float>();
    hd_data->h_q_qd_u[0] = 0.8;
    hd_data->h_q_qd_u[1] = 0.3;
    hd_data->h_q_qd_u[2] = 1;
    hd_data->h_q_qd_u[3] = 0.2;
    hd_data->h_q_qd_u[4] = 0.7;
    hd_data->h_q_qd_u[5] = 0.6;
    hd_data->h_q_qd_u[6] = 0.4;
    hd_data->h_q_qd_u[7] = 0.4;
    hd_data->h_q_qd_u[8] = 0.3;
    hd_data->h_q_qd_u[9] = 0.8;
    hd_data->h_q_qd_u[10] = 0.1;
    
    hd_data->h_q_qd_u[11] = 0;
    hd_data->h_q_qd_u[12] = 0;
    hd_data->h_q_qd_u[13] = 0;
    hd_data->h_q_qd_u[14] = 0;
    hd_data->h_q_qd_u[15] = 0;
    hd_data->h_q_qd_u[16] = 0;
    hd_data->h_q_qd_u[17] = 0;
    hd_data->h_q_qd_u[18] = 0;
    hd_data->h_q_qd_u[19] = 0;
    hd_data->h_q_qd_u[20] = 0;
    hd_data->h_q_qd_u[21] = 0;
    
    hd_data->h_q_qd_u[22] = 0;
    hd_data->h_q_qd_u[23] = 0;
    hd_data->h_q_qd_u[24] = 0;
    hd_data->h_q_qd_u[25] = 0;
    hd_data->h_q_qd_u[26] = 0;
    hd_data->h_q_qd_u[27] = 0;
    hd_data->h_q_qd_u[28] = 0;
    hd_data->h_q_qd_u[29] = 0;
    hd_data->h_q_qd_u[30] = 0;
    hd_data->h_q_qd_u[31] = 0;
    hd_data->h_q_qd_u[32] = 0;


    gpuErrchk(hipMemcpy(hd_data->d_q_qd_u,hd_data->h_q_qd_u,3*grid::NUM_JOINTS*sizeof(float),hipMemcpyHostToDevice));
    gpuErrchk(hipDeviceSynchronize());

    printf("q,qd,u\n");
    printMat<float,1,grid::NUM_JOINTS>(hd_data->h_q_qd_u,1);
    printMat<float,1,grid::NUM_JOINTS>(&hd_data->h_q_qd_u[grid::NUM_JOINTS],1);
    printMat<float,1,grid::NUM_JOINTS>(&hd_data->h_q_qd_u[2*grid::NUM_JOINTS],1);

    printf("aba\n");
    grid::aba<float>(hd_data, d_robotModel, gravity, 1, dim3(1,1,1), dimms, streams);
    printMat<float,1,grid::NUM_JOINTS>(hd_data->h_qdd,1);
    
    return 0;
}

int atlas(){
    grid::gridData<float> *hd_data = grid::init_gridData<float,1>();
    grid::robotModel<float> *d_robotModel = grid::init_robotModel<float>();;
    const int num_timesteps = 1;
    float gravity = static_cast<float>(9.81);
    dim3 dimms(grid::SUGGESTED_THREADS,1,1);
    hipStream_t *streams = grid::init_grid<float>();
    hd_data->h_q_qd_u[0] = 0.8;
    hd_data->h_q_qd_u[1] = 0.3;
    hd_data->h_q_qd_u[2] = 1;
    hd_data->h_q_qd_u[3] = 0.2;
    hd_data->h_q_qd_u[4] = 0.7;
    hd_data->h_q_qd_u[5] = 0.6;
    hd_data->h_q_qd_u[6] = 0.4;
    hd_data->h_q_qd_u[7] = 0.4;
    hd_data->h_q_qd_u[8] = 0.3;
    hd_data->h_q_qd_u[9] = 0.8;
    hd_data->h_q_qd_u[10] = 0.1;
    hd_data->h_q_qd_u[11] = 0.2;
    hd_data->h_q_qd_u[12] = 0.3;
    hd_data->h_q_qd_u[13] = 0.5;
    hd_data->h_q_qd_u[14] = 0.4;
    hd_data->h_q_qd_u[15] = 0.37;
    hd_data->h_q_qd_u[16] = 0.9;
    hd_data->h_q_qd_u[17] = 0.53;
    hd_data->h_q_qd_u[18] = 0.2;
    hd_data->h_q_qd_u[19] = 1;
    hd_data->h_q_qd_u[20] = 0.6;
    hd_data->h_q_qd_u[21] = 0.8;
    hd_data->h_q_qd_u[22] = 0.9;
    hd_data->h_q_qd_u[23] = 0.3;
    hd_data->h_q_qd_u[24] = 0.7;
    hd_data->h_q_qd_u[25] = 0.44;
    hd_data->h_q_qd_u[26] = 0.28;
    hd_data->h_q_qd_u[27] = 0.6;
    hd_data->h_q_qd_u[28] = 1;
    hd_data->h_q_qd_u[29] = 0.1;

    hd_data->h_q_qd_u[60] = 0.1;
    hd_data->h_q_qd_u[61] = 0.1;
    hd_data->h_q_qd_u[62] = 0.1;
    hd_data->h_q_qd_u[63] = 0.1;
    hd_data->h_q_qd_u[64] = 0.1;
    hd_data->h_q_qd_u[65] = 0.1;
    hd_data->h_q_qd_u[66] = 0.1;
    hd_data->h_q_qd_u[67] = 0.1;
    hd_data->h_q_qd_u[68] = 0.1;
    hd_data->h_q_qd_u[69] = 0.1;
    hd_data->h_q_qd_u[70] = 0.1;
    hd_data->h_q_qd_u[71] = 0.1;
    hd_data->h_q_qd_u[72] = 0.1;
    hd_data->h_q_qd_u[73] = 0.1;
    hd_data->h_q_qd_u[74] = 0.1;
    hd_data->h_q_qd_u[75] = 0.1;
    hd_data->h_q_qd_u[76] = 0.1;
    hd_data->h_q_qd_u[77] = 0.1;
    hd_data->h_q_qd_u[78] = 0.1;
    hd_data->h_q_qd_u[79] = 0.1;
    hd_data->h_q_qd_u[80] = 0.1;
    hd_data->h_q_qd_u[81] = 0.1;
    hd_data->h_q_qd_u[82] = 0.1;
    hd_data->h_q_qd_u[83] = 0.1;
    hd_data->h_q_qd_u[84] = 0.1;
    hd_data->h_q_qd_u[85] = 0.1;
    hd_data->h_q_qd_u[86] = 0.1;
    hd_data->h_q_qd_u[87] = 0.1;
    hd_data->h_q_qd_u[88] = 0.1;
    hd_data->h_q_qd_u[89] = 0.1;

    hd_data->h_q_qd_u[30] = 0.1;
    hd_data->h_q_qd_u[31] = 0.1;
    hd_data->h_q_qd_u[32] = 0.1;
    hd_data->h_q_qd_u[33] = 0.1;
    hd_data->h_q_qd_u[34] = 0.1;
    hd_data->h_q_qd_u[35] = 0.1;
    hd_data->h_q_qd_u[36] = 0.1;
    hd_data->h_q_qd_u[37] = 0.1;
    hd_data->h_q_qd_u[38] = 0.1;
    hd_data->h_q_qd_u[39] = 0.1;
    hd_data->h_q_qd_u[40] = 0.1;
    hd_data->h_q_qd_u[41] = 0.1;
    hd_data->h_q_qd_u[42] = 0.1;
    hd_data->h_q_qd_u[43] = 0.1;
    hd_data->h_q_qd_u[44] = 0.1;
    hd_data->h_q_qd_u[45] = 0.1;
    hd_data->h_q_qd_u[46] = 0.1;
    hd_data->h_q_qd_u[47] = 0.1;
    hd_data->h_q_qd_u[48] = 0.1;
    hd_data->h_q_qd_u[49] = 0.1;
    hd_data->h_q_qd_u[50] = 0.1;
    hd_data->h_q_qd_u[51] = 0.1;
    hd_data->h_q_qd_u[52] = 0.1;
    hd_data->h_q_qd_u[53] = 0.1;
    hd_data->h_q_qd_u[54] = 0.1;
    hd_data->h_q_qd_u[55] = 0.1;
    hd_data->h_q_qd_u[56] = 0.1;
    hd_data->h_q_qd_u[57] = 0.1;
    hd_data->h_q_qd_u[58] = 0.1;
    hd_data->h_q_qd_u[59] = 0.1;



    gpuErrchk(hipMemcpy(hd_data->d_q_qd_u,hd_data->h_q_qd_u,3*grid::NUM_JOINTS*sizeof(float),hipMemcpyHostToDevice));
    gpuErrchk(hipDeviceSynchronize());

    printf("q,qd,u\n");
    printMat<float,1,grid::NUM_JOINTS>(hd_data->h_q_qd_u,1);
    printMat<float,1,grid::NUM_JOINTS>(&hd_data->h_q_qd_u[grid::NUM_JOINTS],1);
    printMat<float,1,grid::NUM_JOINTS>(&hd_data->h_q_qd_u[2*grid::NUM_JOINTS],1);

    printf("aba\n");
    grid::aba<float>(hd_data, d_robotModel, gravity, 1, dim3(1,1,1), dimms, streams);
    printMat<float,1,grid::NUM_JOINTS>(hd_data->h_qdd,1);
    
    return 0;
}

int main() {    
    //uncomment the line for the URDF you are using

    iiwa();
    //hyq();
    //hyq_missing_limb();
    // atlas();
    return 0;
}

// int main() {
//     grid::gridData<float> *hd_data = grid::init_gridData<float,1>();
//     grid::robotModel<float> *d_robotModel = grid::init_robotModel<float>();;
//     const int num_timesteps = 1;
//     float gravity = static_cast<float>(9.81);
//     dim3 dimms(grid::SUGGESTED_THREADS,1,1);
//     hipStream_t *streams = grid::init_grid<float>();
//     hd_data->h_q_qd_u[0] = 0.8;
//     hd_data->h_q_qd_u[1] = 0.3;
//     hd_data->h_q_qd_u[2] = 1;
//     hd_data->h_q_qd_u[3] = 0.2;
//     hd_data->h_q_qd_u[4] = 0.7;
//     hd_data->h_q_qd_u[5] = 0.6;
//     hd_data->h_q_qd_u[6] = 0.4;

//     hd_data->h_q_qd_u[7] = 0;
//     hd_data->h_q_qd_u[8] = 0;
//     hd_data->h_q_qd_u[9] = 0;
//     hd_data->h_q_qd_u[10] = 0;
//     hd_data->h_q_qd_u[11] = 0;
//     hd_data->h_q_qd_u[12] = 0;
//     hd_data->h_q_qd_u[13] = 0;

//     hd_data->h_q_qd_u[14] = 0;
//     hd_data->h_q_qd_u[15] = 0;
//     hd_data->h_q_qd_u[16] = 0;
//     hd_data->h_q_qd_u[17] = 0;
//     hd_data->h_q_qd_u[18] = 0;
//     hd_data->h_q_qd_u[19] = 0;
//     hd_data->h_q_qd_u[20] = 0;

//     gpuErrchk(hipMemcpy(hd_data->d_q_qd_u,hd_data->h_q_qd_u,3*grid::NUM_JOINTS*sizeof(float),hipMemcpyHostToDevice));
//     gpuErrchk(hipDeviceSynchronize());

//     printf("q,qd,u\n");
//     printMat<float,1,grid::NUM_JOINTS>(hd_data->h_q_qd_u,1);
//     printMat<float,1,grid::NUM_JOINTS>(&hd_data->h_q_qd_u[grid::NUM_JOINTS],1);
//     printMat<float,1,grid::NUM_JOINTS>(&hd_data->h_q_qd_u[2*grid::NUM_JOINTS],1);

//     printf("aba\n");
//     grid::aba<float>(hd_data, d_robotModel, gravity, 1, dim3(1,1,1), dimms, streams);
//     printMat<float,1,grid::NUM_JOINTS>(hd_data->h_qdd,1);
//     return 0;
// }
